#include "hip/hip_runtime.h"
// Equihash CUDA solver
// Copyright (c) 2016 John Tromp
// Copyright (c) 2018 The ExchangeCoin team

#include "solver_details.cuh"

verify_code equihash_verify_uncompressed(const char *header, u64 header_len, u32 nonce, const proof indices) {
    if (duped(indices))
        return verify_code::POW_DUPLICATE;

    blake2b_state ctx;
    setheader(&ctx, (const uint8_t *)header, header_len, nonce);
    uchar hash[WN / 8];
    return verifyrec(&ctx, indices, hash, WK);
}

verify_code equihash_verify_uncompressed(const std::string &header, u32 nonce, const proof indices) {
    return equihash_verify_uncompressed(header.c_str(), header.length(), nonce, indices);
}

extern "C" int equihash_verify_uncompressed_c(const char *header, u64 header_len, u32 nonce, const proof indices) {
    return static_cast<int>(equihash_verify_uncompressed(header, header_len, nonce, indices));
}

verify_code equihash_verify(const char *header, u64 header_len, u32 nonce, const cproof indices) {
    proof sol;
    uncompress_solution(indices, sol);
    return equihash_verify_uncompressed(header, header_len, nonce, sol);
}

verify_code equihash_verify(const std::string &header, u32 nonce, const cproof indices) {
    return equihash_verify(header.c_str(), header.length(), nonce, indices);
}

extern "C" int equihash_verify_c(const char *header, u64 header_len, u32 nonce, const cproof indices) {
    return static_cast<int>(equihash_verify(header, header_len, nonce, indices));
}

int equihash_solve(const char *header, u64 header_len, u32 nonce, std::function<void(const cproof)> on_solution_found) {
#define printf                                                                                                         \
    if (debug_logs)                                                                                                    \
    printf

    bool debug_logs = false;
    u64 nthreads = 8192;
    u64 tpb = 128; // threads per block, roughly square root of nthreads
    u64 range = 1;

    if (debug_logs) {
        std::string header_hex = to_hex((const unsigned char *)header, header_len);
        printf("Looking for wagner-tree on (\"%s\",%ui", header_hex.c_str(), nonce);
    }

    if (range > 1)
        printf("-%lu", nonce + range - 1);

    printf(") with %d %d-bits digits and %li threads (%li per block)\n", NDIGITS, DIGITBITS, nthreads, tpb);
    equi eq(static_cast<u32>(nthreads));

    u32 *heap0, *heap1;
    checkCudaErrors(hipMalloc((void **)&heap0, sizeof(digit0)));
    checkCudaErrors(hipMalloc((void **)&heap1, sizeof(digit1)));
    for (u32 r = 0; r < WK; r++)
        if ((r & 1) == 0)
            eq.hta.trees0[r / 2] = (bucket0 *)(heap0 + r / 2);
        else
            eq.hta.trees1[r / 2] = (bucket1 *)(heap1 + r / 2);

    checkCudaErrors(hipMalloc((void **)&eq.nslots, 2 * NBUCKETS * sizeof(u32)));
    checkCudaErrors(hipMemset((void *)eq.nslots, 0, 2 * NBUCKETS * sizeof(u32)));
    checkCudaErrors(hipMalloc((void **)&eq.sols, MAXSOLS * sizeof(proof)));

    equi *device_eq;
    checkCudaErrors(hipMalloc((void **)&device_eq, sizeof(equi)));

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    proof sols[MAXSOLS];
    u32 sumnsols = 0;
    for (int r = 0; r < range; r++) {
        hipEventRecord(start, NULL);
        eq.setheadernonce((const uint8_t *)header, header_len, nonce);

        printf("eq.blake_ctx.buf: ");
        for (int i = 0; i < sizeof(eq.blake_ctx.buf); i++)
            printf("%c(%d) ", char(eq.blake_ctx.buf[i]), int(eq.blake_ctx.buf[i]));
        printf("\n");

        checkCudaErrors(hipMemcpy(device_eq, &eq, sizeof(equi), hipMemcpyHostToDevice));
        digitH<<<nthreads / tpb, tpb>>>(device_eq);
        eq.showbsizes(0);
#if BUCKBITS == 16 && RESTBITS == 4 && defined XINTREE && defined(UNROLL)
        digit_1<<<nthreads / tpb, tpb>>>(device_eq);
        eq.showbsizes(1);
        digit2<<<nthreads / tpb, tpb>>>(device_eq);
        eq.showbsizes(2);
        digit3<<<nthreads / tpb, tpb>>>(device_eq);
        eq.showbsizes(3);
        digit4<<<nthreads / tpb, tpb>>>(device_eq);
        eq.showbsizes(4);
        digit5<<<nthreads / tpb, tpb>>>(device_eq);
        eq.showbsizes(5);
        digit6<<<nthreads / tpb, tpb>>>(device_eq);
        eq.showbsizes(6);
        digit7<<<nthreads / tpb, tpb>>>(device_eq);
        eq.showbsizes(7);
        digit8<<<nthreads / tpb, tpb>>>(device_eq);
        eq.showbsizes(8);
#else
        for (u32 r = 1; r < WK; r++) {
            r & 1 ? digitO<<<nthreads / tpb, tpb>>>(device_eq, r) : digitE<<<nthreads / tpb, tpb>>>(device_eq, r);
            eq.showbsizes(r);
        }
#endif
        digitK<<<nthreads / tpb, tpb>>>(device_eq);

        checkCudaErrors(hipMemcpy(&eq, device_eq, sizeof(equi), hipMemcpyDeviceToHost));
        u32 maxsols = min(MAXSOLS, eq.nsols);
        checkCudaErrors(hipMemcpy(sols, eq.sols, maxsols * sizeof(proof), hipMemcpyDeviceToHost));
        hipEventRecord(stop, NULL);
        hipEventSynchronize(stop);
        float duration;
        hipEventElapsedTime(&duration, start, stop);
        printf("%d rounds completed in %.3f seconds.\n", WK, duration / 1000.0f);

        u32 s, nsols, ndupes;
        for (s = nsols = ndupes = 0; s < maxsols; s++) {
            if (duped(sols[s])) {
                ndupes++;
                continue;
            }
            nsols++;
            if (on_solution_found) {
                cproof csol;
                compress_solution(sols[s], csol);
                on_solution_found(csol);
            }
        }
        printf("%d solutions %d dupes\n", nsols, ndupes);
        sumnsols += nsols;
    }
    checkCudaErrors(hipFree(eq.nslots));
    checkCudaErrors(hipFree(eq.sols));
    checkCudaErrors(hipFree(eq.hta.trees0[0]));
    checkCudaErrors(hipFree(eq.hta.trees1[0]));

    printf("%d total solutions\n", sumnsols);

#undef printf
    return 0;
}

int equihash_solve(const std::string &header, u32 nonce, std::function<void(const cproof)> on_solution_found) {
    return equihash_solve(header.c_str(), header.length(), nonce, on_solution_found);
}

extern "C" int equihash_solve_c(const char *header, u64 header_len, u32 nonce,
                                void (*on_solution_found)(void *user_data, const cproof solution), void *user_data) {
    return equihash_solve(header, header_len, nonce,
                          [=](const cproof solution) { on_solution_found(user_data, solution); });
}
