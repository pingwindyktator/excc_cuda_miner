#include <cstdio>
#include <cstdlib>
#include <unistd.h>
#include <iostream>
#include <fstream>
#include <algorithm>
#include "solver.cuh"
#include "test/test.cuh"
#include "test/benchmark.cuh"
#include "utils.hpp"

void onSolutionFound(const u32 *solution) {
    std::cout << "SOLUTION FOUND:\n";

    for (u32 i = 0; i < PROOFSIZE; ++i) {
        std::cout << solution[i] << ' ';
    }
    std::cout << "\n";
    std::string csol = compress_solution(solution);
    std::string solution_hex = to_hex((const unsigned char *)csol.c_str(), csol.length());
    std::cout << solution_hex;


    std::cout << "\n================================================================================================================================\n";
}

int main (int argc, char *argv[]) {
    checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleYield));

    long nthreads = 8192;
    long nonce = 0;
    long tpb = 0;
    long range = 1;
    std::string header;
    int c;

    while ((c = getopt (argc, argv, "h:x:n:r:t:p:T:B")) != -1) {
        switch (c) {
            case 'h':
                header = std::string{optarg};
                break;
            case 'x':
                header = to_bytes(optarg);
                break;
            case 'n':
                nonce = strtol(optarg, nullptr, 10);
                break;
            case 't':
                nthreads = strtol(optarg, nullptr, 10);
                break;
            case 'p':
                tpb = strtol(optarg, nullptr, 10);
                break;
            case 'r':
                range = strtol(optarg, nullptr, 10);
                break;
            case 'T':
                return test();
            case 'B':
                benchmark();
                return 0;
            default:
                break;
        }
    }

    solve(header, nonce, onSolutionFound, nthreads, tpb, range);
}
