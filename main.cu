#include <cstdio>
#include <cstdlib>
#include <unistd.h>
#include <iostream>
#include <fstream>
#include <algorithm>
#include "solver.cuh"
#include "test.cuh"
#include "utils.hpp"

void onSolutionFound(const u32 *solution) {
    std::cout << "SOLUTION FOUND: ";

    for (u32 i = 0; i < PROOFSIZE; ++i) {
        std::cout << solution[i] << ' ';
    }

    std::cout << "\n";
}

int main (int argc, char *argv[]) {
    checkCudaErrors(hipSetDeviceFlags(hipDeviceScheduleYield));

    long nthreads = 8192;
    long nonce = 0;
    long tpb = 0;
    long range = 1;
    std::string header;
    int c;

    while ((c = getopt (argc, argv, "h:x:n:r:t:p")) != -1) {
        switch (c) {
            case 'h':
                header = std::string{optarg};
                break;
            case 'x':
                header = to_bytes(optarg);
                break;
            case 'n':
                nonce = strtol(optarg, nullptr, 10);
                break;
            case 't':
                nthreads = strtol(optarg, nullptr, 10);
                break;
            case 'p':
                tpb = strtol(optarg, nullptr, 10);
                break;
            case 'r':
                range = strtol(optarg, nullptr, 10);
                break;
            default:
                break;
        }
    }

//    check_blocks();
//    test();
    solve(header.c_str(), nonce, onSolutionFound, nthreads, tpb, range);
}
